#include "common.h"

#define CCK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

typedef struct {
    float           sum;
    hipEvent_t     mark;
    char            id[ID_LEN];
} gputimer_t;

void gputimer_start(gputimer_h h)
{
    gputimer_t *t = (gputimer_t*)h;
    CCK(hipEventRecord(t->mark, 0));
}

void gputimer_init(gputimer_h *h, const char *id)
{
    gputimer_t *t = (gputimer_t*)calloc(1, sizeof(gputimer_t));
    strncpy(t->id, id, ID_LEN*sizeof(char));
    CCK(hipEventCreate(&t->mark));
    *h = (gputimer_h)t;
}

void gputimer_pause(gputimer_h h)
{
    gputimer_t *t = (gputimer_t*)h;
    float this_time; hipEvent_t mark; 
    CCK(hipEventCreate(&mark));
    CCK(hipEventRecord(mark, 0));
    CCK(hipEventSynchronize(mark));
    CCK(hipEventElapsedTime(&this_time, t->mark, mark));
    t->sum += this_time;
}

void gputimer_done(gputimer_h h)
{
    gputimer_t *t = (gputimer_t*)h;
    printf(" <%16s>: Elapsed time: %9.1f ms\n", t->id, t->sum);
    CCK(hipEventDestroy(t->mark));
    free(t);
}
