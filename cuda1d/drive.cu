#include <stdio.h>
#include <stdlib.h>
#include <complex.h>
#include <assert.h>
#include <hipfft/hipfft.h>

#define CCK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/**
 * repeat is used to control doing something or just measure overhead
 */
void cuda_fft1d_r2c(hipfftHandle plan, float *pi, int nr, int nmemb, complex float *po, int repeat)
{
    float *d_inp; 
    hipfftComplex *d_out; int nc = nr/2+1;

    CCK(hipMalloc((void**)&d_inp, sizeof(float)*nr*nmemb));
    CCK(hipMalloc((void**)&d_out, sizeof(hipfftComplex)*nr*nmemb));
    CCK(hipMemcpy(d_inp, pi, sizeof(float)*nr*nmemb, hipMemcpyHostToDevice));

    hipfftResult_t code;
    code = hipfftExecR2C(plan, d_inp, d_out);
    assert(code==HIPFFT_SUCCESS);
//  CCK(hipDeviceSynchronize());

    CCK(hipMemcpy(po, d_out, sizeof(hipfftComplex)*nc*nmemb, hipMemcpyDeviceToHost));
    CCK(hipFree(d_inp));
    CCK(hipFree(d_out));
}

hipfftHandle cuda_fft1d_plan(int nr, int howmany)
{
    hipfftHandle plan; int nc=nr/2+1;

    hipfftResult_t code;
    code = hipfftPlanMany(&plan, 1, &nr, &nr, 1, nr, &nc, 1, nc, HIPFFT_R2C, howmany);
    assert(code==HIPFFT_SUCCESS);
    return plan;

}

void cuda_fft1d_destroy(hipfftHandle plan) 
{
    hipfftResult_t code;
    code = hipfftDestroy(plan);
    assert(code==HIPFFT_SUCCESS);
}
