#include "common.h"
#include <hipfft/hipfft.h>

typedef struct {
    hipfftHandle     plan;
    int             fftnr;
    int             rskip;      //
    int             cskip;
    int             howmany;
} cudaplan_t;

#define CCK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/**
 * repeat is used to control doing something or just measure overhead
 */
void cuda1d_r2c(cudaplan_h h, float *pi, complex float *po, int repeat)
{
    float *d_inp; cudaplan_t *t = (cudaplan_t*)h;
    hipfftComplex *d_out; int nc = t->fftnr/2+1;

    CCK(hipMalloc((void**)&d_inp, sizeof(float)*t->fftnr*t->howmany));
    CCK(hipMalloc((void**)&d_out, sizeof(hipfftComplex)*t->fftnr*t->howmany));
    CCK(hipMemcpy(d_inp, pi, sizeof(float)*t->fftnr*t->howmany, hipMemcpyHostToDevice));
    CCK(hipMemset(d_out, 0, sizeof(float)*t->fftnr*t->howmany));

    for(int i=0; i<repeat; i++) {
        hipfftResult_t code = hipfftExecR2C(t->plan, d_inp, d_out);
        assert(code==HIPFFT_SUCCESS);
    }

    CCK(hipMemcpy(po, d_out, sizeof(hipfftComplex)*nc*t->howmany, hipMemcpyDeviceToHost));
    CCK(hipFree(d_inp));
    CCK(hipFree(d_out));
}

void cuda1d_plan(cudaplan_h *h, int fftnr, int rskip, int cskip, int howmany)
{
    cudaplan_t *t; int nc=fftnr/2+1;
    t = (cudaplan_t*)calloc(1, sizeof(cudaplan_t));
    t->fftnr = fftnr;
    t->rskip = rskip;
    t->cskip = cskip;
    t->howmany = howmany;
    hipfftResult_t code = hipfftPlanMany(&t->plan, 1, &fftnr, &fftnr, 1, rskip, &nc, 1, cskip, HIPFFT_R2C, howmany);
    assert(code==HIPFFT_SUCCESS);

    *h = (cudaplan_h)(t);
}

void cuda1d_destroy(cudaplan_h h) 
{
    cudaplan_t *t = (cudaplan_t*)h;
    hipfftResult_t code;
    code = hipfftDestroy(t->plan);
    assert(code==HIPFFT_SUCCESS);
}
