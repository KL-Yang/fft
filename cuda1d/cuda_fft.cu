#include "common.h"
#include <hipfft/hipfft.h>

typedef struct {
    hipfftHandle     plan;
} cudaplan_t;

#define CCK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/**
 * repeat is used to control doing something or just measure overhead
 */
void cuda1d_r2c(cudaplan_h h, float *pi, int nr, int nmemb, complex float *po, int repeat)
{
    float *d_inp; cudaplan_t *t = (cudaplan_t*)h;
    hipfftComplex *d_out; int nc = nr/2+1;

    CCK(hipMalloc((void**)&d_inp, sizeof(float)*nr*nmemb));
    CCK(hipMalloc((void**)&d_out, sizeof(hipfftComplex)*nr*nmemb));
    CCK(hipMemcpy(d_inp, pi, sizeof(float)*nr*nmemb, hipMemcpyHostToDevice));

    for(int i=0; i<repeat; i++) {
        hipfftResult_t code = hipfftExecR2C(t->plan, d_inp, d_out);
        assert(code==HIPFFT_SUCCESS);
    }

    CCK(hipMemcpy(po, d_out, sizeof(hipfftComplex)*nc*nmemb, hipMemcpyDeviceToHost));
    CCK(hipFree(d_inp));
    CCK(hipFree(d_out));
}

void cuda1d_plan(cudaplan_h *h, int nr, int howmany)
{
    cudaplan_t *t; int nc=nr/2+1;
    t = (cudaplan_t*)calloc(1, sizeof(cudaplan_t));

    hipfftResult_t code = hipfftPlanMany(&t->plan, 1, &nr, &nr, 1, nr, &nc, 1, nc, HIPFFT_R2C, howmany);
    assert(code==HIPFFT_SUCCESS);

    *h = (cudaplan_h)(t);
}

void cuda1d_destroy(cudaplan_h h) 
{
    cudaplan_t *t = (cudaplan_t*)h;
    hipfftResult_t code;
    code = hipfftDestroy(t->plan);
    assert(code==HIPFFT_SUCCESS);
}
