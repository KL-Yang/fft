#include "common.h"
#include <hipfft/hipfft.h>

typedef struct {
    hipfftHandle     plan, r2c, c2r;
    int             n;          //fft logical length
    int             skip;       //at least 2*(n/2+1)
    int             howmany;
} cudaplan_t;

#define CCK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/**
 * repeat is used to control doing something or just measure overhead
 */
void cuda1d_r2c(cudaplan_h h, const float *pi, complex float *po, int repeat)
{
    cudaplan_t *t = (cudaplan_t*)h;
    hipfftComplex *d_out; float *d_inp; 

    CCK(hipMalloc((void**)&d_inp, sizeof(float)*t->skip*t->howmany));
    CCK(hipMalloc((void**)&d_out, sizeof(float)*t->skip*t->howmany));
    CCK(hipMemcpy(d_inp, pi, sizeof(float)*t->skip*t->howmany, hipMemcpyHostToDevice));
    CCK(hipMemset(d_out, 0, sizeof(float)*t->skip*t->howmany));

    for(int i=0; i<repeat; i++) {
        hipfftResult_t code = hipfftExecR2C(t->r2c, d_inp, d_out);
        assert(code==HIPFFT_SUCCESS);
    }

    CCK(hipMemcpy(po, d_out, sizeof(float)*t->skip*t->howmany, hipMemcpyDeviceToHost));
    CCK(hipFree(d_inp));
    CCK(hipFree(d_out));
}

/**
 * repeat is used to control doing something or just measure overhead
 */
void cuda1d_c2r(cudaplan_h h, const complex float *pi, float *po, int repeat)
{
    cudaplan_t *t = (cudaplan_t*)h;
    hipfftComplex *d_inp; float *d_out; 

    CCK(hipMalloc((void**)&d_inp, sizeof(float)*t->skip*t->howmany));
    CCK(hipMalloc((void**)&d_out, sizeof(float)*t->skip*t->howmany));
    CCK(hipMemcpy(d_inp, pi, sizeof(float)*t->skip*t->howmany, hipMemcpyHostToDevice));
    CCK(hipMemset(d_out, 0, sizeof(float)*t->skip*t->howmany));

    for(int i=0; i<repeat; i++) {
        hipfftResult_t code = hipfftExecC2R(t->c2r, d_inp, d_out);
        assert(code==HIPFFT_SUCCESS);
    }

    CCK(hipMemcpy(po, d_out, sizeof(float)*t->skip*t->howmany, hipMemcpyDeviceToHost));
    CCK(hipFree(d_inp));
    CCK(hipFree(d_out));
}

/**
 * repeat is used to control doing something or just measure overhead
 */
void cuda1d_c2c(cudaplan_h h, const complex float *pi, complex float *po, int repeat, int flag)
{
    hipfftComplex *d_inp, *d_out;
    cudaplan_t *t = (cudaplan_t*)h;
    int direction = (flag>0)?(HIPFFT_FORWARD):(HIPFFT_BACKWARD);

    CCK(hipMalloc((void**)&d_inp, sizeof(float)*t->skip*t->howmany));
    CCK(hipMalloc((void**)&d_out, sizeof(float)*t->skip*t->howmany));
    CCK(hipMemcpy(d_inp, pi, sizeof(float)*t->skip*t->howmany, hipMemcpyHostToDevice));
    CCK(hipMemset(d_out, 0, sizeof(float)*t->skip*t->howmany));

    for(int i=0; i<repeat; i++) {
        hipfftResult_t code = hipfftExecC2C(t->plan, d_inp, d_out, direction);
        assert(code==HIPFFT_SUCCESS);
    }

    CCK(hipMemcpy(po, d_out, sizeof(float)*t->skip*t->howmany, hipMemcpyDeviceToHost));
    CCK(hipFree(d_inp));
    CCK(hipFree(d_out));
}


/**
 * Note, for r2c and c2r, input/output need padding to the 2*nc float length
 */
void cuda1d_plan(cudaplan_h *h, int n, int skip, int howmany)
{
    cudaplan_t *t = (cudaplan_t*)calloc(1, sizeof(cudaplan_t));
    t->n = n; 
    t->skip = skip; 
    t->howmany = howmany;

    int nr=n, nc=n/2+1; hipfftResult_t code;
    code = hipfftPlanMany(&t->r2c, 1, &nr, &nr, 1, skip, &nc, 1, skip/2, HIPFFT_R2C, howmany);
    assert(code==HIPFFT_SUCCESS);
    code = hipfftPlanMany(&t->c2r, 1, &nr, &nc, 1, skip/2, &nr, 1, skip, HIPFFT_C2R, howmany);
    assert(code==HIPFFT_SUCCESS);
    code = hipfftPlanMany(&t->plan, 1, &n, &n, 1, skip/2, &n, 1, skip/2, HIPFFT_C2C, howmany);
    *h = (cudaplan_h)(t);
}

void cuda1d_destroy(cudaplan_h h) 
{
    hipfftResult_t code;
    cudaplan_t *t = (cudaplan_t*)h;
    code = hipfftDestroy(t->r2c);  assert(code==HIPFFT_SUCCESS);
    code = hipfftDestroy(t->c2r);  assert(code==HIPFFT_SUCCESS);
    code = hipfftDestroy(t->plan); assert(code==HIPFFT_SUCCESS);
}
